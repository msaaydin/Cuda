#include "hip/hip_runtime.h"
#include<stdio.h>
__global__ void add2(int *a)
{
	*a = *a + 285;
}
int main(void)
{
	int *data_h, *data_d;
	hipMalloc((void**)&data_d, sizeof(int));
	data_h = (int *)malloc(sizeof(int));
	*data_h = 15;
	add2 << <1, 1 >> >(data_d);
	hipMemcpy(data_h, data_d, sizeof(int),hipMemcpyDeviceToHost);
	printf("data: %d\n", *data_h);
	free(data_h); hipFree(data_d);
	return 0;
}

/*
Output :

There is no output:
Explanation: data_d points to
uninitialized memory when add2()
is called.

solution:
//hipMemcpy( data_d, data_h, sizeof(int) hipMemcpyHostToDevice );

*/