#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

__global__ void my_first_kernel(float *x)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	x[tid] = (float)threadIdx.x;
}

int main(int argc, char **argv)
{
	float *h_x, *d_x;
	int   nblocks, nthreads, nsize, n;

	nblocks = 2;
	nthreads = 8;
	nsize = nblocks*nthreads;
	h_x = (float *)malloc(nsize*sizeof(float));
	hipMalloc((void **)&d_x, nsize*sizeof(float));
	my_first_kernel << <nblocks, nthreads >> >(d_x);

	hipMemcpy(h_x, d_x, nsize*sizeof(float), hipMemcpyDeviceToHost);

	for (n = 0; n<nsize; n++)
		printf(" n,  x  =  %d  %f \n", n, h_x[n]);

	hipFree(d_x);
	free(h_x);
	return 0;
}
// Output:
/*
n, x = 0  0.0
n, x = 1  1.0
n, x = 2  2.0
n, x = 3  3.0
n, x = 4  4.0
n, x = 5  5.0
n, x = 6  6.0
n, x = 7  7.0
n, x = 8  0.0
n, x = 9  1.0
n, x = 10  2.0
n, x = 11  3.0
n, x = 12  4.0
n, x = 13  5.0
n, x = 14  6.0
n, x = 15  7.0


*/