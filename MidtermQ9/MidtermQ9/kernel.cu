#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

__global__ void my_first_kernel(float *x)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	x[tid] = (float)threadIdx.x;
}

int main(int argc, char **argv)
{
	float *h_x, *d_x;
	int   nblocks, nthreads, nsize, n;

	nblocks = 2;
	nthreads = 8;
	nsize = nblocks*nthreads;
	h_x = (float *)malloc(nsize*sizeof(float));
	hipMalloc((void **)&d_x, nsize*sizeof(float));
	my_first_kernel << <nblocks, nthreads >> >(d_x);

	hipMemcpy(h_x, d_x, nsize*sizeof(float), hipMemcpyDeviceToHost);

	for (n = 0; n<nsize; n++)
		printf(" n,  x  =  %d  %f \n", n, h_x[n]);

	hipFree(d_x);
	free(h_x);
	return 0;
}
