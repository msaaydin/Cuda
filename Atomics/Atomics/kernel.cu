#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "gputimer.h"

#define NUM_THREADS 1000000
#define ARRAY_SIZE  100

#define BLOCK_WIDTH 1000

void print_array(int *array, int size)
{
	printf("{ ");
	for (int i = 0; i < size; i++)  { printf("%d ", array[i]); }
	printf("}\n");
}

__global__ void increment_naive(int *g)
{
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;
	g[i] = g[i] + 1;
}

__global__ void increment_atomic(int *g)
{
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;
	atomicAdd(&g[i], 1);
}

int main(int argc, char **argv)
{
	GpuTimer timer;
	printf("%d total threads in %d blocks writing into %d array elements\n",
		NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

	
	int h_array[ARRAY_SIZE];
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	
	int * d_array;
	hipMalloc((void **)&d_array, ARRAY_BYTES);
	hipMemset((void *)d_array, 0, ARRAY_BYTES);

	// launch the kernel - comment out one of these
	timer.Start();
	// increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
	increment_atomic <<<NUM_THREADS / BLOCK_WIDTH, BLOCK_WIDTH >>>(d_array);
	timer.Stop();

	
	hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
	print_array(h_array, ARRAY_SIZE);
	printf("Time elapsed = %g ms\n", timer.Elapsed());

	
	hipFree(d_array);
	return 0;
}