
#include "hip/hip_runtime.h"

#define thread_size 128;
#include <stdio.h>

const int  N = 400;


// CUDA Kernel for Vector Addition
__global__ void Vector_Addition(const int *dev_a, const int *dev_b, int *dev_c)
{
	//Get the id of thread within a block	
	unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < N) // check the boundry condition for the threads
		dev_c[tid] = dev_a[tid] + dev_b[tid];
}


int main(void)
{
		//Host array
		int Host_a[N], Host_b[N], Host_c[N];

		//Device array
		int *dev_a, *dev_b, *dev_c;
		int block_size = N / thread_size;
		//Allocate the memory on the GPU
		hipMalloc((void **)&dev_a, N*sizeof(int));
		hipMalloc((void **)&dev_b, N*sizeof(int));
		hipMalloc((void **)&dev_c, N*sizeof(int));

		//fill the Host array with random elements on the CPU
		for (int i = 0; i <N; i++)
		{
			Host_a[i] = i+2;
			Host_b[i] = i*i;
		}

		//Copy Host array to Device array
		hipMemcpy(dev_a, Host_a, N*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, Host_b, N*sizeof(int), hipMemcpyHostToDevice);

		//Make a call to GPU kernel
		Vector_Addition <<< block_size, 128>>> (dev_a, dev_b, dev_c);

		//Copy back to Host array from Device array
		hipMemcpy(Host_c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

		//Display the result
		for (int i = 0; i<N; i++)
			printf("%d + %d = %d\n", Host_a[i], Host_b[i], Host_c[i]);

		//Free the Device array memory
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);		
		return 0;

}



