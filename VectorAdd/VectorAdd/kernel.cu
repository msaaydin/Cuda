
#include "hip/hip_runtime.h"

#define thread_size 128
#include <stdio.h>
#include <math.h>

double  N = 400;
const int size = 4000;

// CUDA Kernel for Vector Addition
__global__ void Vector_Addition(const int *dev_a, const int *dev_b, int *dev_c)
{
	//Get the id of thread within a block	
	unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < size) // check the boundry condition for the threads
	    dev_c[tid] = dev_a[tid] + dev_b[tid];
	

}

int main(void)
{
		//Host array
		int Host_a[size], Host_b[size], Host_c[size];

		//Device array
		int *dev_a, *dev_b, *dev_c;
		int block_size =(int)((N / thread_size)+0.99);
		//Allocate the memory on the GPU
		hipMalloc((void **)&dev_a, size*sizeof(int));
		hipMalloc((void **)&dev_b, size*sizeof(int));
		hipMalloc((void **)&dev_c, size*sizeof(int));

		//fill the Host array with random elements on the CPU
		for (int i = 0; i <size; i++)
		{
			Host_a[i] = i+2;
			Host_b[i] = i*i;
		}

		//Copy Host array to Device array
		hipMemcpy(dev_a, Host_a, size*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, Host_b, size*sizeof(int), hipMemcpyHostToDevice);

		//Make a call to GPU kernel
		Vector_Addition <<< block_size, 128>>> (dev_a, dev_b, dev_c);

		//Copy back to Host array from Device array
		hipMemcpy(Host_c, dev_c, size*sizeof(int), hipMemcpyDeviceToHost);

		//Display the result
		for (int i = 0; i<size; i++)
			printf("%d + %d = %d\n", Host_a[i], Host_b[i], Host_c[i]);
		//printf("%d + %d = %d",Host_a[400],Host_b[400],Host_c[400]);
		//Free the Device array memory
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);		
		return 0;

}



