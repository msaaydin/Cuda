
#include "hip/hip_runtime.h"

#define thread_size 128
#include <stdio.h>
#include <math.h>
const long N = 16 * 16;


// CUDA Kernel for Vector Addition
__global__ void Vector_Addition( long *dev_a)
{
	//Get the id of thread within a block	
	unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < N) // check the boundry condition for the threads
		dev_a[tid] = dev_a[tid] + 45;

}

int main(void)
{
	//Host array
	long Host_a[N];

	//Device array
	long *dev_a;
	long block_size = (int)((N / thread_size) + 0.99);
	//Allocate the memory on the GPU
	hipMalloc((void **)&dev_a, N*sizeof(long));
	
	//fill the Host array with random elements on the CPU
	for (long i = 0; i <N; i++)
	{
		Host_a[i] = i + 2;
	}
	for (int i = 0; i<100; i++)
		printf(" = %d\n", Host_a[i]);
	printf("************************************************\n");

	//Copy Host array to Device array
	hipMemcpy(dev_a, Host_a, N*sizeof(long), hipMemcpyHostToDevice);
	
	//Make a call to GPU kernel
	Vector_Addition <<< block_size, 128 >>> (dev_a);

	//Copy back to Host array from Device array
	hipMemcpy(Host_a, dev_a, N*sizeof(long), hipMemcpyDeviceToHost);

	//Display the result
	for (int i = 0; i<100; i++)
		printf(" =%d \n", Host_a[i]);
	//printf("%d + %d = %d",Host_a[400],Host_b[400],Host_c[400]);
	//Free the Device array memory
	hipFree(dev_a);
	return 0;

}



