#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>

//#define N 512
//#define threadsize 512

#include <stdio.h>
#include <stdlib.h>

#define N 8
#define M 8
#define A(i,j) A[(i)*cols+(j)]  // row-major layout
#define B(i,j) B[(i)*cols+(j)]  // row-major layout
#define C(i,j) C[(i)*cols+(j)]  // row-major layout

__global__ void MatAdd(const double *A, const double *B, double *C,
	int rows, int cols)
{
	int row = threadIdx.y;
	int col = threadIdx.x;
	if ((row < rows) && (col < cols)) {
		C(row, col) = A(row, col) + B(row, col);
	}
}

int main(void)
{
	double A[N][M] = { { 1, 2, 3, 4 },
					   { 5, 6, 7, 8 },
					   { 9, 0, 1, 2 } };
	double B[N][M] = { { 3, 3, 3, 3 },
					   { 1, 1, 1, 1 },
	                   { 0, 0, 0, 0 } };
	double *C;
	double *A_d = 0, *B_d = 0, *C_d = 0;
	int rows = N;
	int cols = M;
	dim3 blockDim(M, N);
	C = (double *)malloc(sizeof(*C)*N*M);
	hipMalloc((void**)&A_d, sizeof(*A_d)*N*M);
	hipMalloc((void**)&B_d, sizeof(*B_d)*N*M);
	hipMalloc((void**)&C_d, sizeof(*C_d)*N*M);
	hipMemcpy(A_d, A, sizeof(*A_d)*N*M, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, sizeof(*B_d)*N*M, hipMemcpyHostToDevice);
	MatAdd << <1, blockDim >> >(A_d, B_d, C_d, rows, cols);
	hipMemcpy(C, C_d, sizeof(*C)*N*M, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			printf("%g ", C(i, j));
		}
		printf("\n");
	}
	hipFree(C_d);
	hipFree(B_d);
	hipFree(A_d);
	free(C);
	return EXIT_SUCCESS;
}
/*__global__ void matrixAdd(int *a, int *b, int *c) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = col + row * N;
	if (col < N && row < N) {

		c[index] = a[index] + b[index];
	}
}


int main() {
	int a[N][N], b[N][N], c[N][N];
	int *dev_a, *dev_b, *dev_c;
	int size = N * N * sizeof(int);
	for (int i = 0; i < N; i++)
	{ 
		for (int j = 0; j < N; j++)
		{
			a[i][j] = i + j;
			b[i][j] = i * j;

		}

	}



	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	int blockSize = size / N;
	
	matrixAdd <<<blockSize,threadsize>>>(dev_a, dev_b, dev_c);
	hipDeviceSynchronize();

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; i++)
	{
		for (int j = 0; j < 10; j++)
		{
			printf("%d ", a[i][j]);
			
		}
		printf("\n");
	}

	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
}*/
