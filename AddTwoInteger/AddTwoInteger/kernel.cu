
#include "hip/hip_runtime.h"


#include <stdio.h>

	__global__ void add(int *a, int *b,int *c)
	{
		*c = *a + *b;
	}

int main()
{
    
	int a, b, c; // host copies of a, b, c
	int *d_a, *d_b, *d_c;  // device copies of a, b, c
	int	size =sizeof(int);// Allocate space for device copies of a, b, c

	hipMalloc((void**)&d_a, size);

	hipMalloc((void**)&d_b, size);

	hipMalloc((void**)&d_c, size);
	// Setup input values
	a = 26754;
	b = 73456;

	// Copy inputs to device
	hipMemcpy(d_a, &a, size,hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size,hipMemcpyHostToDevice);
	// Launch add() kernel on GPU

	add <<<1, 1 >>>(d_a,d_b,d_c);

	// Copy result back to host
	hipMemcpy(&c,d_c, size,hipMemcpyDeviceToHost);
	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	printf("toplam sonucu : %d\n", c);
	
	return 0;
}


